#include "hip/hip_runtime.h"
/*
 * Tanner Hoelzel
 */

#include <random>
#include <cmath>

#include "io.h"

#define gpu_assert(rv) gpu_assert_h((rv), __FILE__, __LINE__)
void
gpu_assert_h(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

class Network {
public:
    float *inputs;
    float *weights;
    float *hidden;
    float *weights2;
    float *outputs;
    float *classes;
    float *loss;
};

float *
random_weights(unsigned int n_weights) {
    float *weights = (float *)malloc(n_weights*sizeof(float));
    std::default_random_engine eng(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for(unsigned int i = 0; i < n_weights; i++) {
        weights[i] = dist(eng);
    }
    return weights;
}

__global__ void
hidden_layer(float *input, float *weights, float *output) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;

    weights = weights + id*28*28;

    float dp = 0;
    for (unsigned int i = 0; i < 28*28; i++) {
        dp += weights[i]*input[i];
    }
    output[id] = dp > 0? dp : 0; //ReLU
}

/*
void
softmax_layer(float *input, float *output) {
    for(int class = 0; class < 10; class++) {
        float max = 0;
        for(int i = 0; i < 1024; i++) {
            if(input[i] > max) max = input[i];
        }
        float sum = 0;
        for(int i = 0; i < 1024; i++) {
            output[i] = std::exp(in[i] - C);
        }
    }
}

    T sum = 0;
    for (size_t i = 0; i < N; i++) {
        out[i] = std::exp(in[i] - C);
        sum += out[i];
    }
    std::transform(out.begin(), out.end(), out.begin(), [sum](auto e) { return e/sum; });

    // Verify that it is a probability: Sums to 1 and all >= 0.
    assert(approx_equal(std::accumulate(out.begin(), out.end(), T(0)), 1));
    #ifndef NDEBUG
    std::for_each(out.begin(), out.end(), [](auto e) { assert(e >= 0); });
    #endif

    return out;
}
*/


void
train(float *images, char *labels, float *weights) {

}

void
test(float *images, char *labels, float *weights) {

    float *hidden;
    hipMalloc(&hidden, 1024*sizeof(float));

    hidden_layer<<<1, 1024>>>(images, weights, hidden);
    //gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());

    hipFree(hidden);
}

/*
 * Read inputs, labels, and weights into GPU
 */
int main(int argc, char **argv) {

    char *labels, *test_labels;
    float *images, *test_images, *weights;

    labels = read_labels("mnist/train-labels-idx1-ubyte", 60000);
    test_labels = read_labels("mnist/t10k-labels-idx1-ubyte", 10000);
    images = read_images("mnist/train-images-idx3-ubyte", 60000);
    test_images = read_images("mnist/t10k-images-idx3-ubyte", 10000);
    weights = random_weights(28*28*1024*sizeof(float));

    char *d_labels, *d_test_labels;
    float *d_images, *d_test_images, *d_weights;

    hipMalloc(&d_labels, 60000*sizeof(char));
    hipMalloc(&d_test_labels, 10000*sizeof(char));
    hipMalloc(&d_images, 60000*28*28*sizeof(float));
    hipMalloc(&d_test_images, 10000*28*28*sizeof(float));
    hipMalloc(&d_weights, 28*28*1024*sizeof(float));

    hipMemcpy(d_labels, labels, 60000*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_test_labels, test_labels, 10000*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_images, images, 60000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_test_images, test_images, 10000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, 28*28*1024*sizeof(float), hipMemcpyHostToDevice);

    Network net;
    
    for(int e = 0; e < 1; e++) {
        train(d_images, d_labels, d_weights);
        test(d_test_images, d_test_labels, d_weights);
    }

    hipFree(d_labels);
    hipFree(d_test_labels);
    hipFree(d_images);
    hipFree(d_test_images);
    hipFree(d_weights);

    free(labels);
    free(test_labels);
    free(images);
    free(test_images);
    free(weights);
}
