/*
 * Tanner Hoelzel
 */


#include <hip/hip_runtime.h>
#include <random>

extern "C" float * read_images(char *, unsigned int);
extern "C" char * read_labels(char *, unsigned int);

float *
random_weights(unsigned int n_weights) {
    float *weights = (float *)malloc(n_weights*sizeof(float));
    std::default_random_engine eng(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for(unsigned int i = 0; i < n_weights; i++) {
        weights[i] = dist(eng);
    }
    return weights;
}

void
train(float *images, char *labels, float *weights) {

}

/*
 * Read inputs, labels, and weights into GPU
 */
int main(int argc, char **argv) {

    char *labels, *test_labels;
    float *images, *test_images, *weights;

    labels = read_labels("mnist/train-labels-idx1-ubyte", 60'000);
    test_labels = read_labels("mnist/t10k-labels-idx1-ubyte", 10'000);
    images = read_images("mnist/train-images-idx3-ubyte", 60'000);
    test_images = read_images("mnist/t10k-images-idx3-ubyte", 10'000);
    weights = random_weights(28*28*1024*sizeof(float));

    char *d_labels, *d_test_labels;
    float *d_images, *d_test_images, *d_weights;

    hipMalloc(&d_labels, 60'000*sizeof(char));
    hipMalloc(&d_test_labels, 10'000*sizeof(char));
    hipMalloc(&d_images, 60'000*28*28*sizeof(float));
    hipMalloc(&d_test_images, 10'000*28*28*sizeof(float));
    hipMalloc(&d_weights, 28*28*1024*sizeof(float));

    hipMemcpy(d_labels, labels, 60'000*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_test_labels, test_labels, 10'000*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_images, images, 60'000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_test_images, test_images, 10'000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, 28*28*1024*sizeof(float), hipMemcpyHostToDevice);

    train(images, labels, weights);

    hipFree(d_labels);
    hipFree(d_test_labels);
    hipFree(d_images);
    hipFree(d_test_images);
    hipFree(d_weights);

    free(labels);
    free(test_labels);
    free(images);
    free(test_images);
    free(weights);
}
