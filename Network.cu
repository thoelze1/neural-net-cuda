#include "hip/hip_runtime.h"
/*
 * Tanner Hoelzel
 */

#include <cmath>
#include <iostream>
#include <algorithm>
#include <vector>

#include "Network.h"

#define N_NODES       1024
#define BATCH_SIZE     100
#define RATE          0.01

Network::Network(float *inputs, unsigned char *labels) {

    float weights1[28*28*N_NODES];
    float weights2[N_NODES*10];

    this->eng = new std::default_random_engine(std::random_device{}());

    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for(unsigned int i = 0; i < 28*28*N_NODES; i++) {
        weights1[i] = dist(*(this->eng));
    }
    for(unsigned int i = 0; i < N_NODES*10; i++) {
        weights2[i] = dist(*(this->eng));
    }

    this->host_labels = labels;
    hipMalloc(&this->inputs, 28*28*60000*sizeof(float));
    hipMalloc(&this->weights1, 28*28*N_NODES*sizeof(float));
    hipMalloc(&this->outputs, N_NODES*sizeof(float));
    hipMalloc(&this->weights2, N_NODES*10*sizeof(float));
    hipMalloc(&this->classes, 10*sizeof(float));
    hipMalloc(&this->softmax, 10*sizeof(float));
    hipMalloc(&this->softmax_ds, 10*sizeof(float));
    hipMalloc(&this->hidden_ds, 1024*sizeof(float));
    hipMalloc(&this->weights1_grad, 28*28*1024*sizeof(float));
    hipMalloc(&this->weights2_grad, 1024*10*sizeof(float));

    hipMemcpy(this->inputs, inputs, 60000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->weights1, weights1, 28*28*1024*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->weights2, weights2, 1024*10*sizeof(float), hipMemcpyHostToDevice);
}

Network::~Network() {
    hipFree(this->inputs);
    hipFree(this->weights1);
    hipFree(this->outputs);
    hipFree(this->weights2);
    hipFree(this->classes);
    hipFree(this->softmax);
    delete this->eng;
}

__global__ void
softmax_forward(float *input, float *output, unsigned int n) {
    unsigned int i;
    float max = 0;
    for(i = 0; i < n; i++) {
        if(input[i] > max) max = input[i];
    }
    float sum = 0;
    for(i = 0; i < n; i++) {
        output[i] = exp(input[i] - max);
        sum += output[i];
    }
    for(i = 0; i < n; i++) {
        output[i] = output[i]/sum;
    }
}

__global__ void
softmax_back(float *sm_out, unsigned char label, float *sm_ds) {

    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    float us = -1/sm_out[label];
    if (id == (unsigned int)label) {
        sm_ds[id] = (sm_out[label]*(1 - sm_out[id]))*us;
    } else {
        sm_ds[id] = (-1*sm_out[id]*sm_out[label])*us;
    }
}

__global__ void
hidden_forward(float *input, unsigned int input_size, float *weights, float *output, unsigned int output_size, bool relu) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;
    weights = weights + id*input_size;

    float dp = 0;
    for (unsigned int i = 0; i < input_size; i++) {
        dp += weights[i]*input[i];
    }
    output[id] = (!relu || dp > 0)? dp : 0;
}

//TODO use multiple blocks to make this faster
__global__ void
hidden_back(float *hidden_out, float *sm_ds, float *hidden_ds, float *weights2, float *weights2_grad, float *layer) {

    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    for(unsigned int i = 0; i < 10; i++) {
        hidden_ds[id] += sm_ds[i]*weights2[id*10+i];
        weights2_grad[id*10+i] += sm_ds[i]*layer[id];
    }
}

void
Network::run(unsigned int i) {

    hidden_forward<<<1, 1024>>>(this->inputs+i*28*28, 28*28, this->weights1, this->outputs, 1024, true);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());

    hidden_forward<<<1, 10>>>(this->outputs, 1024, this->weights2, this->classes, 10, false);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());

    softmax_forward<<<1, 1>>>(this->classes, this->softmax, 10);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());

    softmax_back<<<1, 10>>>(this->softmax, this->host_labels[i], this->softmax_ds);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());

    hidden_back<<<1, 1024>>>(this->classes, this->softmax_ds, this->hidden_ds, this->weights2, this->weights2_grad, this->outputs);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());
}

void
Network::update() {

}

void
Network::train() {
    std::vector<unsigned int> indices(60000);
    for(unsigned int i = 0; i < 60000; i++) {
        indices[i] = i;
    }
    std::shuffle(std::begin(indices), std::end(indices), *(this->eng));
    /*
    for(unsigned int i = 0; i < (60000/BATCH_SIZE); i++) {
        hipMemset(this->weights1_grad, 0, 28*28*1024*sizeof(float))
        hipMemset(this->weights2_grad, 0, 28*28*1024*sizeof(float))
        for(unsigned int j = 0; j < BATCH_SIZE; j++) {
            run(indices[i*BATCH_SIZE+j]);
        }
        update();
    }
    */
    run(0);
}

float
Network::test(float *tests, unsigned char *labels) {

    float *d_tests;
    hipMalloc(&d_tests, 28*28*10000*sizeof(float));
    hipMemcpy(d_tests, tests, 28*28*10000*sizeof(float), hipMemcpyHostToDevice);

    unsigned int acc = 0;
    //@TODO change back to 10000
    for(unsigned int i = 0; i < 1000; i++) {
        hidden_forward<<<1, 1024>>>(d_tests + i*28*28, 28*28, this->weights1, this->outputs, 1024, true);
        gpu_assert(hipPeekAtLastError());
        gpu_assert(hipDeviceSynchronize());
        hidden_forward<<<1, 10>>>(this->outputs, 1024, this->weights2, this->classes, 10, false);
        gpu_assert(hipPeekAtLastError());
        gpu_assert(hipDeviceSynchronize());
        float mem[10];
        hipMemcpy(mem, this->classes, 10*sizeof(float), hipMemcpyDeviceToHost);
        float max = -100000;
        unsigned int max_j = 0;
        for(unsigned int j = 0; j < 10; j++) {
            if(mem[i] > max) {
                max = mem[j];
                max_j = j;
            }
        }
        if(((int)labels[i]) == max_j) acc += 1;
    }
    return (float)acc/1000;
}

