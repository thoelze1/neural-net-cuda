/*
 * Tanner Hoelzel
 */

#include "Network.h"

Network::~Network() {
    hipFree(this->inputs);
    hipFree(this->labels);
    hipFree(this->weights1);
    hipFree(this->weights2);
    hipFree(this->scratch);
}
