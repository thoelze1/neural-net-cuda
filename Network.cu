#include "hip/hip_runtime.h"
/*
 * Tanner Hoelzel
 */

#include <cmath>
#include <iostream>
#include <algorithm>
#include <vector>

#include "Network.h"

#define N_NODES       1024
#define BATCH_SIZE     600
#define RATE         0.002
#define DO_RATE        0.4
#define IMG_SIZE     28*28

Network::Network(float *inputs, unsigned char *labels) {

    float input_w[IMG_SIZE*N_NODES];
    float hidden_w[N_NODES*10];

    this->eng = new std::default_random_engine(std::random_device{}());

    std::normal_distribution<float> dist;
    for(unsigned int i = 0; i < IMG_SIZE*N_NODES; i++) {
        input_w[i] = dist(*(this->eng))/sqrt(IMG_SIZE);
    }
    for(unsigned int i = 0; i < N_NODES*10; i++) {
        hidden_w[i] = dist(*(this->eng))/sqrt(N_NODES);
    }

    this->host_labels = labels;
    hipMalloc(&this->input_l, IMG_SIZE*60000*sizeof(float));
    hipMalloc(&this->input_w, IMG_SIZE*N_NODES*sizeof(float));
    hipMalloc(&this->input_w_grad, IMG_SIZE*N_NODES*sizeof(float));
    hipMalloc(&this->input_bias, N_NODES*sizeof(float));
    hipMalloc(&this->input_bias_grad, N_NODES*sizeof(float));
    hipMalloc(&this->hidden_l, N_NODES*sizeof(float));
    hipMalloc(&this->hidden_w, N_NODES*10*sizeof(float));
    hipMalloc(&this->hidden_w_grad, N_NODES*10*sizeof(float));
    hipMalloc(&this->hidden_bias, 10*sizeof(float));
    hipMalloc(&this->hidden_bias_grad, 10*sizeof(float));
    hipMalloc(&this->dropouts, N_NODES*sizeof(float));
    hipMalloc(&this->output_l, 10*sizeof(float));
    hipMalloc(&this->softmax_l, 10*sizeof(float));
    hipMalloc(&this->softmax_ds, 10*sizeof(float));
    hipMalloc(&this->hidden_ds, N_NODES*sizeof(float));

    hipMemset(this->input_bias, 0, N_NODES*sizeof(float));
    hipMemset(this->hidden_bias, 0, 10*sizeof(float));

    hipMemcpy(this->input_l, inputs, 60000*IMG_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->input_w, input_w, IMG_SIZE*N_NODES*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->hidden_w, hidden_w, N_NODES*10*sizeof(float), hipMemcpyHostToDevice);
}

Network::~Network() {
    hipFree(this->input_l);
    hipFree(this->input_w);
    hipFree(this->input_w_grad);
    hipFree(this->input_bias);
    hipFree(this->input_bias_grad);
    hipFree(this->hidden_l);
    hipFree(this->hidden_w);
    hipFree(this->hidden_w_grad);
    hipFree(this->hidden_bias);
    hipFree(this->hidden_bias_grad);
    hipFree(this->output_l);
    hipFree(this->softmax_l);
    hipFree(this->softmax_ds);
    hipFree(this->hidden_ds);
    delete this->eng;
}

__global__ void
softmax_forward(float *input, float *output, unsigned int n) {
    unsigned int i;
    float max = input[0];
    for(i = 1; i < n; i++) {
        if(input[i] > max) max = input[i];
    }
    float sum = 0;
    for(i = 0; i < n; i++) {
        output[i] = expf(input[i] - max);
        sum += output[i];
    }
    for(i = 0; i < n; i++) {
        output[i] = output[i]/sum;
    }
}

__global__ void
softmax_back(float *softmax_l, float *softmax_ds, unsigned char label) {

    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    float us = -1/softmax_l[(unsigned int)label];
    if (id == (unsigned int)label) {
        softmax_ds[id] = (softmax_l[(unsigned int)label]*(1 - softmax_l[id]))*us;
    } else {
        softmax_ds[id] = -1*softmax_l[id]*softmax_l[(unsigned int)label]*us;
    }
}

__global__ void
hidden_forward(float *input, unsigned int input_size, float *weights, float *output, unsigned int output_size,
               float *bias, bool relu, float *dropouts) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;

    float dp = 0;
    for (unsigned int i = 0; i < input_size; i++) {
        dp += weights[id*input_size+i]*input[i];
    }
    dp += bias[id];
    if(dropouts) {
        dp *= dropouts[id];
    }
    output[id] = (!relu || (dp > 0))? dp : 0;
}

//TODO use multiple blocks to make this faster
__global__ void
hidden_back(float *input, unsigned int input_size, float *output, unsigned int output_size,
            float *us, float *ds, float *weights, float *weights_grad, float *bias, float *bias_grad,
            bool relu) {

    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    for(unsigned int i = 0; i < output_size; i++) {
        if(!relu || output[i] > 0) {
            if(ds) {
                ds[id] += us[i]*weights[id*output_size+i];
            }
            weights_grad[id*output_size+i] += (us[i]*input[id]/BATCH_SIZE);
        }
        if(id == 0) {
            bias_grad[i] += us[i]/BATCH_SIZE;
        }
    }
}

__global__ void
update_weights(float *weights, float *weights_grad) {
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    weights[id] -= weights_grad[id]*RATE;
}

void
Network::train(unsigned int i) {

    float dropouts[N_NODES];
    std::uniform_real_distribution<float> dist;
    for(unsigned int i = 0; i < N_NODES; i++) {
        if(dist(*this->eng) < DO_RATE) dropouts[i] = 0;
        else dropouts[i] = 1/DO_RATE;
    }
    hipMemcpy(this->dropouts, dropouts, N_NODES*sizeof(float), hipMemcpyHostToDevice);

    hipMemset(this->hidden_ds, 0, N_NODES*sizeof(float));

    hidden_forward<<<1, N_NODES>>>(this->input_l+i*IMG_SIZE, IMG_SIZE, this->input_w, this->hidden_l, N_NODES,
                                this->input_bias, true, 0);
    hidden_forward<<<1, 10>>>(this->hidden_l, N_NODES, this->hidden_w, this->output_l, 10,
                              this->hidden_bias, false, this->dropouts);
    softmax_forward<<<1, 1>>>(this->output_l, this->softmax_l, 10);
    softmax_back<<<1, 10>>>(this->softmax_l, this->softmax_ds, this->host_labels[i]);
    hidden_back<<<1, N_NODES>>>(this->hidden_l, N_NODES, this->output_l, 10,
                             this->softmax_ds, this->hidden_ds, this->hidden_w, this->hidden_w_grad,
                             this->hidden_bias, this->hidden_bias_grad, false);
    hidden_back<<<1, IMG_SIZE>>>(this->input_l+i*IMG_SIZE, IMG_SIZE, this->hidden_l, N_NODES,
                              this->hidden_ds, 0, this->input_w, this->input_w_grad,
                              this->input_bias, this->input_bias_grad, true);

    float mem[10];
    std::cout << (unsigned int)this->host_labels[i] << std::endl;
    hipMemcpy(mem, this->softmax_l, 10*sizeof(float), hipMemcpyDeviceToHost);
    for(unsigned int j = 0; j < 10; j++) {
        std::cout << mem[j] << " ";
    }
    std::cout << std::endl;
}

void
Network::train() {
    std::vector<unsigned int> indices(60000);
    for(unsigned int i = 0; i < 60000; i++) {
        indices[i] = i;
    }
    std::shuffle(std::begin(indices), std::end(indices), *(this->eng));
    for(unsigned int i = 0; i < (60000/BATCH_SIZE); i++) {
        std::cout << "Batch " << i << std::endl;
        hipMemset(this->input_w_grad, 0, IMG_SIZE*N_NODES*sizeof(float));
        hipMemset(this->input_bias_grad, 0, N_NODES*sizeof(float));
        hipMemset(this->hidden_w_grad, 0, N_NODES*10*sizeof(float));
        hipMemset(this->hidden_bias_grad, 0, 10*sizeof(float));
        for(unsigned int j = 0; j < BATCH_SIZE; j++) {
            train(indices[i*BATCH_SIZE+j]);
        }
        update_weights<<<IMG_SIZE, N_NODES>>>(this->input_w, this->input_w_grad);
        update_weights<<<N_NODES, 10>>>(this->hidden_w, this->hidden_w_grad);
        update_weights<<<1, N_NODES>>>(this->input_bias, this->input_bias_grad);
        update_weights<<<1, 10>>>(this->hidden_bias, this->hidden_bias_grad);
    }
}

float
Network::test(float *tests, unsigned char *labels) {

    float *d_tests;
    hipMalloc(&d_tests, IMG_SIZE*10000*sizeof(float));
    hipMemcpy(d_tests, tests, IMG_SIZE*10000*sizeof(float), hipMemcpyHostToDevice);

    unsigned int acc = 0;
    for(unsigned int i = 0; i < 1000; i++) {
        hidden_forward<<<1, N_NODES>>>(d_tests + i*IMG_SIZE, IMG_SIZE, this->input_w, this->hidden_l, N_NODES, this->input_bias, true, 0);
        hidden_forward<<<1, 10>>>(this->hidden_l, N_NODES, this->hidden_w, this->output_l, 10, this->hidden_bias, false, 0);
        float mem[10];
        hipMemcpy(mem, this->output_l, 10*sizeof(float), hipMemcpyDeviceToHost);
        float max = mem[0];
        unsigned int max_j = 0;
        for(unsigned int j = 0; j < 10; j++) {
            if(mem[j] > max) {
                max = mem[j];
                max_j = j;
            }
        }
        if(((unsigned int)labels[i]) == max_j) acc += 1;
    }
    return (float)acc/1000;
}
