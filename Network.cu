#include "hip/hip_runtime.h"
/*
 * Tanner Hoelzel
 */

#include <cmath>
#include <iostream>
#include <algorithm>
#include <vector>

#include "Network.h"

#define N_NODES       1024
#define BATCH_SIZE     100

Network::Network(float *inputs, unsigned char *labels) {

    float weights1[28*28*N_NODES];
    float weights2[N_NODES*10];

    this->eng = new std::default_random_engine(std::random_device{}());

    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for(unsigned int i = 0; i < 28*28*N_NODES; i++) {
        weights1[i] = dist(*(this->eng));
    }
    for(unsigned int i = 0; i < N_NODES*10; i++) {
        weights2[i] = dist(*(this->eng));
    }

    hipMalloc(&this->labels, 60000*sizeof(char));
    hipMalloc(&this->inputs, 28*28*60000*sizeof(float));
    hipMalloc(&this->weights1, 28*28*N_NODES*sizeof(float));
    hipMalloc(&this->outputs, N_NODES*sizeof(float));
    hipMalloc(&this->weights2, N_NODES*10*sizeof(float));
    hipMalloc(&this->classes, 10*sizeof(float));
    hipMalloc(&this->softmax, 10*sizeof(float));

    hipMemcpy(this->labels, labels, 60000*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(this->inputs, inputs, 60000*28*28*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->weights1, weights1, 28*28*1024*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->weights2, weights2, 1024*10*sizeof(float), hipMemcpyHostToDevice);
}

Network::~Network() {
    hipFree(this->labels);
    hipFree(this->inputs);
    hipFree(this->weights1);
    hipFree(this->outputs);
    hipFree(this->weights2);
    hipFree(this->classes);
    hipFree(this->softmax);
    delete this->eng;
}

__global__ void
forward(float *input, unsigned int input_size, float *weights, float *output, unsigned int output_size, bool relu) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;
    weights = weights + id*input_size;

    float dp = 0;
    for (unsigned int i = 0; i < input_size; i++) {
        dp += weights[i]*input[i];
    }
    output[id] = (!relu || dp > 0)? dp : 0;
}

__global__ void
softmax_forward(float *input, float *output, unsigned int n) {
    unsigned int i;
    float max = 0;
    for(i = 0; i < n; i++) {
        if(input[i] > max) max = input[i];
    }
    float sum = 0;
    for(i = 0; i < n; i++) {
        output[i] = exp(input[i] - max);
        sum += output[i];
    }
    for(i = 0; i < n; i++) {
        output[i] = output[i]/sum;
    }
}

void
Network::run(unsigned int index) {
    forward<<<1, 1024>>>(this->inputs, 28*28, this->weights1, this->outputs, 1024, true);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());
    forward<<<1, 10>>>(this->outputs, 1024, this->weights2, this->classes, 10, false);
    gpu_assert(hipPeekAtLastError());
    gpu_assert(hipDeviceSynchronize());
    softmax_forward<<<1, 1>>>(this->classes, this->softmax, 10);
    /*
    float mem[10];
    hipMemcpy(mem, this->softmax, 10*sizeof(float), hipMemcpyDeviceToHost);
    float sum = 0;
    for(unsigned int i = 0; i < 10; i++) {
        sum += mem[i];
    }
    std::cout << sum << std::endl;
    */
}

void
Network::train() {
    std::vector<unsigned int> indices(60000);
    for(unsigned int i = 0; i < 60000; i++) {
        indices[i] = i;
    }
    std::shuffle(std::begin(indices), std::end(indices), *(this->eng));
    for(unsigned int i = 0; i < (60000/BATCH_SIZE); i++) {
        for(unsigned int j = 0; j < BATCH_SIZE; j++) {
            //run(indices[i*BATCH_SIZE+j]);
        }
    }
}

float
Network::test(float *tests, unsigned char *labels) {

    float *d_tests;
    hipMalloc(&d_tests, 28*28*10000*sizeof(float));
    hipMemcpy(d_tests, tests, 28*28*10000*sizeof(float), hipMemcpyHostToDevice);

    unsigned int acc = 0;
    for(unsigned int i = 0; i < 10000; i++) {
        forward<<<1, 1024>>>(d_tests + i*28*28, 28*28, this->weights1, this->outputs, 1024, true);
        gpu_assert(hipPeekAtLastError());
        gpu_assert(hipDeviceSynchronize());
        forward<<<1, 10>>>(this->outputs, 1024, this->weights2, this->classes, 10, false);
        gpu_assert(hipPeekAtLastError());
        gpu_assert(hipDeviceSynchronize());
        float mem[10];
        hipMemcpy(mem, this->classes, 10*sizeof(float), hipMemcpyDeviceToHost);
        float max = -100000;
        unsigned int max_j = 0;
        for(unsigned int j = 0; j < 10; j++) {
            if(mem[i] > max) {
                max = mem[j];
                max_j = j;
            }
        }
        if(((int)labels[i]) == max_j) acc += 1;
    }
    return (float)acc/10000;
}

