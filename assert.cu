/*
 * Tanner Hoelzel
 */


#include <hip/hip_runtime.h>
#include <cstdio>

void
gpu_assert_h(hipError_t code, const char *file, int line, bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
